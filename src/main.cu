#include <cstdio>
#include <cassert>

#include "cxxopts.hpp"

#include "table.hpp"

int main(int argc, char **argv) {

        std::string output_format;
        bool print_descriptions;

        try
        {
          cxxopts::Options options(argv[0], " - format CUDA device info");
          options
            .positional_help("[optional args]")
            .show_positional_help();
      
          options
        //     .allow_unrecognised_options()
            .add_options()
            ("f,format", "Output Format", cxxopts::value<std::string>(output_format)->default_value("shell"), "FMT")
            ("d,descriptions", "Print Descriptions", cxxopts::value<bool>(print_descriptions)->default_value("false"))
            ("h,help", "Print help")
          ;
      
          auto result = options.parse(argc, argv);
      
          if (result.count("help"))
          {
            std::cout << options.help({"", "Group"}) << std::endl;
            exit(0);
          }
      
      
        } catch (const cxxopts::OptionException& e)
        {
          std::cout << "error parsing options: " << e.what() << std::endl;
          exit(1);
      }

    int err = 0;
    int n;
    hipGetDeviceCount(&n);

    for (int i = 0; i < n; ++i) {

        Table table;
        table.Header(0) = "Property";
        table.Header(1) = "Value";
        if (print_descriptions) {
                table.Header(2) = "Description";
        }


        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        table.Titlef("Device %d: %s", i, prop.name);

#if __CUDACC_VER_MAJOR__ > 8 && __CUDACC_VER_MINOR__ >  1
        table.NewRow();
        table.Cell("hipDeviceProp_t.pageableMemoryAccessUsesHostPageTables");
        table.Cellf("%d", prop.pageableMemoryAccessUsesHostPageTables);
        if (print_descriptions) {
                table.Cell("Device accesses pageable memory using host page tables. This suggests Address Translation Services are enabled on Power9");
        }
#endif

#if __CUDACC_VER_MAJOR__ > 8
        table.NewRow();
        table.Cell("hipDeviceProp_t.canUseHostPointerForRegisteredMem");
        table.Cellf("%d", prop.canUseHostPointerForRegisteredMem);
        if (print_descriptions) {
                table.Cell("Device can access host registered memory at the same virtual address as the CPU");
        }
#endif

#if __CUDACC_VER_MAJOR__ > 8 && __CUDACC_VER_MINOR__ >  1
{
        int v;
        hipDeviceGetAttribute ( &v, hipDeviceAttributeDirectManagedMemAccessFromHost, i );
        table.NewRow();
        table.Cell("hipDeviceAttributeDirectManagedMemAccessFromHost");
        table.Cellf("%d", v);
        if (print_descriptions) {
                table.Cell("Host can directly access managed memory on the device without migration");
        }
}
#endif

#if __CUDACC_VER_MAJOR__ > 8 && __CUDACC_VER_MINOR__ >  1
{
        int v;
        hipDeviceGetAttribute ( &v, cudaDevAttrCanFlushRemoteWrites, i );
        table.NewRow();
        table.Cell("cudaDevAttrCanFlushRemoteWrites");
        table.Cellf("%d", v);
        if (print_descriptions) {
                table.Cell("device supports flushing of outstanding remote writes");
        }
}
#endif

        table.NewRow();
        table.Cell("hipDeviceProp_t.pageableMemoryAccess");
        table.Cellf("%d", prop.pageableMemoryAccess);
        if (print_descriptions) {
                table.Cell("Device supports coherently accessing pageable memory without calling hipHostRegister on it.");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.concurrentManagedAccess");
        table.Cellf("%d", prop.concurrentManagedAccess);
        if (print_descriptions) {
                table.Cell("Device can coherently access managed memory concurrently with the CPU.");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.canMapHostMemory");
        table.Cellf("%d", prop.canMapHostMemory);
        if (print_descriptions) {
                table.Cell("Device can map host memory into the CUDA address space for use with hipHostAlloc()/hipHostGetDevicePointer()");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.totalGlobalMem");
        table.Cellf("%lu", prop.totalGlobalMem);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.totalConstMem");
        table.Cellf("%lu", prop.totalConstMem);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.sharedMemPerBlock");
        table.Cellf("%lu", prop.sharedMemPerBlock);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.sharedMemPerMultiprocessor");
        table.Cellf("%lu", prop.sharedMemPerMultiprocessor);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.l2CacheSize");
        table.Cellf("%d", prop.l2CacheSize);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.memoryBusWidth");
        table.Cellf("%d", prop.memoryBusWidth);
        if (print_descriptions) {
                table.Cell("bits");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.memoryClockRate");
        table.Cellf("%d", prop.memoryClockRate);
        if (print_descriptions) {
                table.Cell("kHz");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.asyncEngineCount");
        table.Cellf("%d", prop.asyncEngineCount);
        if (print_descriptions) {
                table.Cell("1: concurrent kernel and copy, 2: kernel and duplex copy");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.globalL1CacheSupported");
        table.Cellf("%d", prop.globalL1CacheSupported);
        if (print_descriptions) {
                table.Cell("Device supports caching of globals in L1 cache");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.localL1CacheSupported");
        table.Cellf("%d", prop.localL1CacheSupported);
        if (print_descriptions) {
                table.Cell("Device supports caching of locals in L1 cache");
        }

        {
                hipSetDevice(i);
                hipSharedMemConfig config;
                hipDeviceGetSharedMemConfig ( &config );
                if (hipSharedMemBankSizeFourByte == config) {
                        printf("\tcudaDeviceGetSharedMemConfig: hipSharedMemBankSizeFourByte\n");
                } else if ( hipSharedMemBankSizeEightByte == config) {
                        printf("\tcudaDeviceGetSharedMemConfig: hipSharedMemBankSizeEightByte\n");
                } else {
                        printf("\tcudaDeviceGetSharedMemConfig: UNKNOWN\n");
                        err = 1;
                }
        }

        {
                hipSetDevice(i);
                hipFuncCache_t config;
                hipDeviceGetCacheConfig ( &config );
                if (hipFuncCachePreferNone == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferNone\n");
                        printf("\t\tno preference for shared memory or L1, or sizes are fixed\n");
                } else if ( hipFuncCachePreferShared == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferShared\n");
                        printf("\t\tprefer larger shared memory and smaller L1 cache\n");
                } else if ( hipFuncCachePreferL1 == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferL1\n");
                        printf("\t\tprefer larger L1 cache and smaller shared memory\n");
                } else if ( hipFuncCachePreferEqual == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferEqual\n");
                        printf("\t\tprefer equal size L1 cache and shared memory\n");
                } else {
                        printf("\tcudaDeviceGetCacheConfig: UNKNOWN\n");
                        err = 1;
                }
        }

        printf("%s\n", table.csv_str().c_str());
        printf("%s\n", table.md_str().c_str());
        printf("%s\n", table.shell_str().c_str());

    }



    return err;
}