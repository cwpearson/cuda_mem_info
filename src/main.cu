#include <cstdio>
#include <cassert>

#include "cxxopts.hpp"

#include "table.hpp"
#include "report.hpp"



//for ioctls
// #include <sys/ioctl.h>
// #include <nvidia-uvm/uvm_ioctl.h>
// #include <nv.h>
// for files
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

// #undef XAPIGEN
// #undef NV_STATUS_CODE
// #define NV_STATUS_CODE( name, code, string ) { code,  string " [" #name "]" },

// static struct NvStatusCodeString
// {
//     NV_STATUS   statusCode;
//     const char *statusString;
// } g_StatusCodeList[] = {
//    #include <nvstatuscodes.h>
//    { 0xffffffff, "Unknown error code!" } // Some compilers don't like the trailing ','
// };

// const char *nvstatusToString(NV_STATUS nvStatusIn)
// {
//     NvU32 i;
//     NvU32 n = ((NvU32)(sizeof(g_StatusCodeList))/(NvU32)(sizeof(g_StatusCodeList[0])));
//     printf("%d\n", n);
//     for (i = 0; i < 10; i++)
//     {
//         printf("%d\n", g_StatusCodeList[i].statusCode);
//         if (g_StatusCodeList[i].statusCode == nvStatusIn)
//         {
//             return g_StatusCodeList[i].statusString;
//         }
//     }

//     return "Unknown error code!";
// }

int main(int argc, char **argv) {

        std::string output_format;
        bool print_descriptions;

        Report report("CUDA Properties");

        try
        {
          cxxopts::Options options(argv[0], "Format CUDA device info");
          options
            .positional_help("[optional args]")
            .show_positional_help();
      
          options
        //     .allow_unrecognised_options()
            .add_options()
            ("f,format", "Output Format", cxxopts::value<std::string>(output_format)->default_value("shell"), "shell|csv|md")
            ("d,descriptions", "Print Descriptions", cxxopts::value<bool>(print_descriptions)->default_value("false"))
            ("h,help", "Print help")
          ;
      
          auto result = options.parse(argc, argv);
      
          if (result.count("help"))
          {
            std::cout << options.help({"", "Group"}) << std::endl;
            exit(0);
          }
      
      
        } catch (const cxxopts::OptionException& e)
        {
          std::cout << "error parsing options: " << e.what() << std::endl;
          exit(1);
      }

      /*
      auto path = "/dev/nvidia-uvm";
      auto fd = open(path, O_RDWR);
      if (fd == -1)
        {
        perror("open");
        return 2;
        }

      UVM_PAGEABLE_MEM_ACCESS_PARAMS q;

      if (ioctl(fd, UVM_PAGEABLE_MEM_ACCESS, &q) == -1)
      {
          perror("ioctl get");
      }
      else
      {
          printf("pageableMemAccess : %d\n", q.pageableMemAccess);
          printf("rmStatus: %d\n", q.rmStatus);
      }
      close(fd);

      fd = open("/dev/nvidia0", O_RDWR);
      if (fd == -1)
        {
        perror("open");
        return 2;
        }
    */

/*
      nv_ioctl_numa_info_t numa_info;
      numa_info.nid = 0;
      if (ioctl(fd, NV_ESC_NUMA_INFO, &numa_info) == -1)
      {
          perror("ioctl get");
      }
      else
      {
        printf("%d\n", numa_info.nid);
        printf("%d\n", numa_info.status);
        printf("%llu\n", numa_info.memblock_size);
        printf("%llu\n", numa_info.numa_mem_addr);
        printf("%llu\n", numa_info.numa_mem_size);
        // nv_blacklist_addresses_t blacklist_addresses NV_ALIGN_BYTES(8);
      }
*/
/*
        nv_ioctl_card_info_t info;
        void *pinfo = &info;
        nv_ioctl_xfer_t xfer;
        xfer.cmd = NV_ESC_CARD_INFO;
        xfer.size = 65536;
        xfer.ptr = &pinfo;

        nv_ioctl_rm_api_version_t ver;

      if (ioctl(fd, NV_ESC_CHECK_VERSION_STR, &ver) == -1)
      {
          perror("ioctl get");
      }
      else
      {
        printf("%d\n", ver.reply);
      }
      
      exit(0);
*/
    int err = 0;
    int n;
    hipGetDeviceCount(&n);

    for (int i = 0; i < n; ++i) {

        Table table;
        table.Header(0) = "Property";
        table.Header(1) = "Value";
        if (print_descriptions) {
                table.Header(2) = "Description";
        }


        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        table.Titlef("Device %d: %s", i, prop.name);

#if __CUDACC_VER_MAJOR__ > 8 && __CUDACC_VER_MINOR__ >  1
        table.NewRow();
        table.Cell("hipDeviceProp_t.pageableMemoryAccessUsesHostPageTables");
        table.Cellf("%d", prop.pageableMemoryAccessUsesHostPageTables);
        if (print_descriptions) {
                table.Cell("Device accesses pageable memory using host page tables. This suggests Address Translation Services are enabled on Power9");
        }
#endif

#if __CUDACC_VER_MAJOR__ > 8
        table.NewRow();
        table.Cell("hipDeviceProp_t.canUseHostPointerForRegisteredMem");
        table.Cellf("%d", prop.canUseHostPointerForRegisteredMem);
        if (print_descriptions) {
                table.Cell("Device can access host registered memory at the same virtual address as the CPU");
        }
#endif

#if __CUDACC_VER_MAJOR__ > 8 && __CUDACC_VER_MINOR__ >  1
{
        int v;
        hipDeviceGetAttribute ( &v, hipDeviceAttributeDirectManagedMemAccessFromHost, i );
        table.NewRow();
        table.Cell("hipDeviceAttributeDirectManagedMemAccessFromHost");
        table.Cellf("%d", v);
        if (print_descriptions) {
                table.Cell("Host can directly access managed memory on the device without migration");
        }
}
#endif

#if __CUDACC_VER_MAJOR__ > 8 && __CUDACC_VER_MINOR__ >  1
{
        int v;
        hipDeviceGetAttribute ( &v, cudaDevAttrCanFlushRemoteWrites, i );
        table.NewRow();
        table.Cell("cudaDevAttrCanFlushRemoteWrites");
        table.Cellf("%d", v);
        if (print_descriptions) {
                table.Cell("device supports flushing of outstanding remote writes");
        }
}
#endif

        table.NewRow();
        table.Cell("hipDeviceProp_t.pageableMemoryAccess");
        table.Cellf("%d", prop.pageableMemoryAccess);
        if (print_descriptions) {
                table.Cell("Device supports coherently accessing pageable memory without calling hipHostRegister on it.");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.concurrentManagedAccess");
        table.Cellf("%d", prop.concurrentManagedAccess);
        if (print_descriptions) {
                table.Cell("Device can coherently access managed memory concurrently with the CPU.");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.canMapHostMemory");
        table.Cellf("%d", prop.canMapHostMemory);
        if (print_descriptions) {
                table.Cell("Device can map host memory into the CUDA address space for use with hipHostAlloc()/hipHostGetDevicePointer()");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.totalGlobalMem");
        table.Cellf("%lu", prop.totalGlobalMem);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.totalConstMem");
        table.Cellf("%lu", prop.totalConstMem);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.sharedMemPerBlock");
        table.Cellf("%lu", prop.sharedMemPerBlock);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.sharedMemPerMultiprocessor");
        table.Cellf("%lu", prop.sharedMemPerMultiprocessor);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.l2CacheSize");
        table.Cellf("%d", prop.l2CacheSize);
        if (print_descriptions) {
                table.Cell("bytes");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.memoryBusWidth");
        table.Cellf("%d", prop.memoryBusWidth);
        if (print_descriptions) {
                table.Cell("bits");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.memoryClockRate");
        table.Cellf("%d", prop.memoryClockRate);
        if (print_descriptions) {
                table.Cell("kHz");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.asyncEngineCount");
        table.Cellf("%d", prop.asyncEngineCount);
        if (print_descriptions) {
                table.Cell("1: concurrent kernel and copy, 2: kernel and duplex copy");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.globalL1CacheSupported");
        table.Cellf("%d", prop.globalL1CacheSupported);
        if (print_descriptions) {
                table.Cell("Device supports caching of globals in L1 cache");
        }

        table.NewRow();
        table.Cell("hipDeviceProp_t.localL1CacheSupported");
        table.Cellf("%d", prop.localL1CacheSupported);
        if (print_descriptions) {
                table.Cell("Device supports caching of locals in L1 cache");
        }

        {
                hipSetDevice(i);
                hipSharedMemConfig config;
                hipDeviceGetSharedMemConfig ( &config );
                if (hipSharedMemBankSizeFourByte == config) {
                        printf("\tcudaDeviceGetSharedMemConfig: hipSharedMemBankSizeFourByte\n");
                } else if ( hipSharedMemBankSizeEightByte == config) {
                        printf("\tcudaDeviceGetSharedMemConfig: hipSharedMemBankSizeEightByte\n");
                } else {
                        printf("\tcudaDeviceGetSharedMemConfig: UNKNOWN\n");
                        err = 1;
                }
        }

        {
                hipSetDevice(i);
                hipFuncCache_t config;
                hipDeviceGetCacheConfig ( &config );
                if (hipFuncCachePreferNone == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferNone\n");
                        printf("\t\tno preference for shared memory or L1, or sizes are fixed\n");
                } else if ( hipFuncCachePreferShared == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferShared\n");
                        printf("\t\tprefer larger shared memory and smaller L1 cache\n");
                } else if ( hipFuncCachePreferL1 == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferL1\n");
                        printf("\t\tprefer larger L1 cache and smaller shared memory\n");
                } else if ( hipFuncCachePreferEqual == config) {
                        printf("\tcudaDeviceGetCacheConfig: hipFuncCachePreferEqual\n");
                        printf("\t\tprefer equal size L1 cache and shared memory\n");
                } else {
                        printf("\tcudaDeviceGetCacheConfig: UNKNOWN\n");
                        err = 1;
                }
        }

        if ("shell" == output_format) {
                printf("%s\n", report.ascii_str().c_str());
                printf("%s\n", table.shell_str().c_str());
        } else if ("csv" == output_format) {
                printf("%s\n", table.csv_str().c_str());
        } else if ("md" == output_format) {
                printf("%s\n", table.md_str().c_str());
        } else {
                fprintf(stderr, "unexpected output value: %s\n", output_format.c_str());
                return 1;
        }

    }



    return err;
}